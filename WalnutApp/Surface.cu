#include "hip/hip_runtime.h"
#include "Mat.cuh"
#include "Surface.cuh"
#include "Vec3.cuh"

namespace RTTrace {
	__host__ bool init_bound(SurfaceInfo& surface) {
		AABB& bound = surface.bound;
		switch (surface.type) {
		case SurfaceInfo::PLANE:
			bound.active = false;
			return false;
		case SurfaceInfo::SPHERE:
		{
			Vec3 unit = Vec3(surface.scale, surface.scale, surface.scale);
			bound.minw = surface.origin - unit;
			bound.maxw = surface.origin + unit;
			bound.active = true;
		}
			return true;
		case SurfaceInfo::TRIANGLE:
			bound.minw = surface.points[0];
			bound.maxw = surface.points[0];
			for (int p = 1; p < 3; p++) {
				for (int axis = 0; axis < 3; axis++) {
					bound.minw[axis] = std::min(surface.points[p][axis], bound.minw[axis]);
					bound.maxw[axis] = std::max(surface.points[p][axis], bound.maxw[axis]);
				}
			}
			bound.active = true;
			return true;
		}
	}

	__device__ bool hit_bound(const Ray& r, const SurfaceInfo& surface) {
		const AABB& bound = surface.bound;
		const Vec3& ro = r.origin;
		const Vec3& rd = r.dir;
		if (!bound.active) return true;
		float tming = T_EPSILON, tmaxg = T_MAX;
		for (int axis = 0; axis < 3; axis++) {
			float tmin = (bound.minw[axis] - ro[axis]) / rd[axis];
			float tmax = (bound.maxw[axis] - ro[axis]) / rd[axis];
			if (tmin > tmax) {
				float t = tmin;
				tmin = tmax;
				tmax = t;
			}
			tming = fmaxf(tmin, tming);
			tmaxg = fminf(tmax, tmaxg);
			if (tmaxg < tming) return false;
		}
		return true;
	}

	__device__ bool hit_sphere (const Ray& ray, const SurfaceInfo& surface, HitInfo& hit) {
		Vec3 origin = surface.origin;
		float radius = surface.scale;

		Vec3 rd = norm(ray.dir);
		Vec3 ro = ray.origin - origin;
		float a = 1.0;
		float b = 2.0 * dot(rd, ro);
		float c = dot(ro, ro) - radius*radius;

		float discSquared = b * b - 4 * a * c;
		if (discSquared < 0) return false;
		float t_small = (-b - sqrt(discSquared)) / (2 * a);
		float t_big = (-b + sqrt(discSquared)) / (2 * a);
		if (t_small > T_MAX || t_big < T_EPSILON) return false;
		float t = t_small < T_EPSILON ? t_big : t_small;
		hit.is_hit = true;
		hit.t = t;
		hit.view_dir = rd;
		hit.pos = ray.point(t); // dir must be unit vector
		hit.norm = norm(hit.pos - origin);
		return true;
	}
	__device__ bool hit_plane (const Ray& ray, const SurfaceInfo& surface, HitInfo& hit) {
		Vec3 normal = norm(surface.normal);
		Vec3 d = norm(ray.dir);
		float n_dot_dir = dot(normal, d);
		if (n_dot_dir == 0) return false; // ray parallel to plane
		float t = dot(normal, surface.origin - ray.origin) / n_dot_dir;
		if (t < T_EPSILON || t > T_MAX) return false;
		hit.is_hit = true;
		hit.t = t;
		hit.view_dir = d;
		hit.pos = ray.point(t); // dir must be unit vector
		hit.norm = normal;
		return true;	
	}

	__device__ bool hit_triangle(const Ray& ray, const SurfaceInfo& surface, HitInfo& hit) {
		const Vec3& rd = ray.dir;
		const Vec3& ro = ray.origin;
		Vec3& ab = surface.points[1] - surface.points[0];
		Vec3& ac = surface.points[2] - surface.points[0];
		Vec3 normal = norm(cross(ac, ab));
		// Naive geometric method (anti-clockwise handedness), faster than Moller-Trombore here
		// Check intersection with the plane
		/*
		float n_dot_dir = dot(normal, rd);
		if (n_dot_dir == 0) return false; // ray parallel to plane
		float t = dot(normal, surface.points[0] - ray.origin) / n_dot_dir;
		if (t < T_EPSILON || t > T_MAX) return false;
		Vec3 p = ray.point(t);
		Vec3 bc = surface.points[2] - surface.points[1];
		Vec3 ca = surface.points[0] - surface.points[2];
		if (dot(cross(ab, p - surface.points[0]), normal) < 0 || 
			dot(cross(bc, p - surface.points[1]), normal) < 0 ||
			dot(cross(-ac, p - surface.points[2]), normal) < 0) return false;
		*/

		// Moller-Trombore (uses Cramer's rule): Benchmark: 7 FPS
		Vec3 aro = ro - surface.points[0];
		Vec3 d = -rd;
		float detA = det(d, ab, ac);
		float t = det(aro, ab, ac) / detA;
		float beta = det(d, aro, ac) / detA;
		float gamma = det(d, ab, aro) / detA;
		if (t < T_EPSILON || t > T_MAX) return false;
		if (beta < 0 || gamma < 0 || beta + gamma > 1) return false;
		Vec3 p = surface.points[0] * (1 - beta - gamma) + surface.points[1] * beta + surface.points[2] * gamma;
		hit.is_hit = true;
		hit.t = t;
		hit.pos = p;
		hit.view_dir = rd;
		hit.norm = normal;
		return true;
	}

}
