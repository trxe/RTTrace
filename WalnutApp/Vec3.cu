#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Vec3.cuh"
#include "Utils.cuh"

namespace RTTrace {
	__host__ __device__ Vec3 operator-(const Vec3& u) {
		return Vec3(-u[0], -u[1], -u[2]);
	}

	__host__ __device__ float dot(const Vec3& u, const Vec3& v) {
		return u[0] * v[0] + u[1] * v[1] + u[2] * v[2];
	}

	__host__ __device__ Vec3 reflect(const Vec3& l, const Vec3& n) {
		return n * dot(n, l) * 2.0f - l;
	}

	__host__ __device__ Vec3 norm(const Vec3& u) {
		float l = u.len();
		if (l == 0) return u;
		return u / u.len();
	}

	__host__ __device__ float det(const Vec3& c0, const Vec3& c1, const Vec3& c2) {
		float d = 0;
		d += c0[0] * (c1[1] * c2[2] - c1[2] * c2[1]);
		d -= c1[0] * (c0[1] * c2[2] - c0[2] * c2[1]);
		d += c2[0] * (c0[1] * c1[2] - c0[2] * c1[1]);
		return d;
	}

	__host__ __device__ Vec3 cross(const Vec3 &u, const Vec3 &v) {
		return Vec3(u[1] * v[2] - u[2] * v[1],
			u[2] * v[0] - u[0] * v[2],
			u[0] * v[1] - u[1] * v[0]);
	}

	__host__ __device__ Vec3 clamp(const Vec3& u, const Vec3& min, const Vec3& max) {
		Vec3 uclamp = u;
		for (int i = 0; i < 3; i++) {
			uclamp[i] = fminf(max[i], uclamp[i]);
			uclamp[i] = fmaxf(min[i], uclamp[i]);
		}
		return uclamp;
	}

	__host__ __device__ Vec3 vmin(const Vec3& u, const Vec3& v) {
		Vec3 omin = u;
		for (int i = 0; i < 3; i++) {
			omin[i] = fminf(u[i], v[i]);
		}
		return omin;
	}

	__host__ __device__ Vec3 vmax(const Vec3& u, const Vec3& v) {
		Vec3 omax = u;
		for (int i = 0; i < 3; i++) {
			omax[i] = fmaxf(u[i], v[i]);
		}
		return omax;
	}

	/**
	 * Clamps vector to a color ranging between WHITE and BLACK.
	 * 
	 * \param u Unclamped Vector
	 * \return Color after clamping
	 */
	__host__ __device__ Vec3 clamp_color(const Vec3& u) {
		return clamp(u, Vec3(), Vec3(1.0, 1.0, 1.0));
	}

	/**
	 * Finds the Vec3 of parameters t yielding the interpolated u between min and max.
	 * 
	 * \param u Resultant interpolated vector
	 * \param min Min vector
	 * \param max Max vector
	 * \return Vec3 of parameters such that lerp(t, min, max) is u
	 */
	__host__ __device__ Vec3 inverse_lerp(const Vec3& u, const Vec3& min, const Vec3& max) {
		Vec3 t{};
		for (int i = 0; i < 3; i++) {
			if (min[i] == max[i]) t[i] = NAN;
			t[i] = (u[i] - min[i]) / (max[i] - min[i]);
		}
		return t;
	}

	/**
	 * Generates morton code required for LBVH generation.
	 * 
	 * \param pos World space coordinates of centroid of object.
	 * \param global_min World space coordinates of the minimum point on the global AABB.
	 * \param global_max World space coordinates of the maximum point on the global AABB.
	 */
	__host__ __device__ abgr_t vec3_to_abgr(const Vec3 &u) {
		abgr_t result;
		Vec3 uclamp = clamp_color(u);
		int r = (int)(uclamp[0] * 255.0);
		int g = (int)(uclamp[1] * 255.0);
		int b = (int)(uclamp[2] * 255.0);
		result = 0xff000000 | (abgr_t)b << 16 | (abgr_t)g << 8 | (abgr_t)r;
		return result;
	}

	std::ostream& operator<<(std::ostream& os, const Vec3& vec) {
		os << "{";
		for (int i = 0; i < 3; i++) {
			os << vec[i] << " ";
		}
		os << "}";
		return os;
	}

	__host__ __device__ uint32_t generate_morton_code(const Vec3& pos, const Vec3& global_min, const Vec3& global_max) {
		uint32_t morton = 0x0;
		Vec3 rel_pos = inverse_lerp(pos, global_min, global_max);
		for (int i = 0; i < 3; i++) {
			uint32_t v = fminf(fmaxf(0.0, rel_pos[i] * 1024.0), 1023.0);
			uint32_t vv = left_shift_3(v);
			morton += vv << (2 - i);
		}
		return morton;
	}
}