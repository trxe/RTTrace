#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Vec3.cuh"
#include "Utils.cuh"

namespace RTTrace {
	__host__ __device__ float dot(const Vec3& u, const Vec3& v) {
		return u[0] * v[0] + u[1] * v[1] + u[2] * v[2];
	}

	__host__ __device__ Vec3 norm(const Vec3& u) {
		return u / u.len();
	}

	__host__ __device__ Vec3 cross(const Vec3 &u, const Vec3 &v) {
		return Vec3(u[1] * v[2] - u[2] * v[1],
			u[2] * v[0] - u[0] * v[2],
			u[0] * v[1] - u[1] * v[0]);
	}

	__host__ __device__ abgr_t vec3_to_abgr(const Vec3 &u) {
		abgr_t result;
		Vec3 unorm = norm(u);
		int r = (int)(fabs(unorm[0]) * 256.0);
		int g = (int)(fabs(unorm[1]) * 256.0);
		int b = (int)(fabs(unorm[2]) * 256.0);
		result = 0xff000000 | b << 16 | g << 8 | r;
		return result;
	}

	std::ostream& operator<<(std::ostream& os, const Vec3& vec) {
		os << "{";
		for (int i = 0; i < 3; i++) {
			os << vec[i] << " ";
		}
		os << "}";
		return os;
	}
}